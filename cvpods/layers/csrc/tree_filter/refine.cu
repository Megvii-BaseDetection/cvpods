#include "hip/hip_runtime.h"
/* Copyright (C) 2019-2021 Megvii Inc. All rights reserved. */
#include <math.h>
#include <thread>
#include <vector>
#include <deque>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#define CUDA_NUM_THREADS         64
#define GET_CUDA_CHANNEL(N)      ceil(512.0f / N)

__global__ void root_leaf_prop_kernel(
        float * in_data,
        float * out_data,
        float * weight,
        int * sorted_index,
        int * sorted_parent_index,
        int batch_size,
        int channel_size,
        int vertex_count){

    const int thread_idx    = threadIdx.x;
    const int batch_idx     = blockIdx.x;
    const int channel_idx   = blockIdx.y;
    const int thread_count  = blockDim.x;
    const int channel_step  = gridDim.y;

    in_data             += batch_idx * vertex_count * channel_size;
    out_data            += batch_idx * vertex_count * channel_size;
    weight              += batch_idx * vertex_count;
    sorted_index        += batch_idx * vertex_count;
    sorted_parent_index += batch_idx * vertex_count;

    __shared__ int node_per_thread[CUDA_NUM_THREADS];
    node_per_thread[thread_idx] = -1;
    if (thread_idx == 0){
        weight[0]              = 0;
        sorted_parent_index[0] = 0;
    }
    __syncthreads();

    int i = thread_idx;
    while (i < vertex_count){
        int par = sorted_parent_index[i];
        int par_thread = par % thread_count;
        if ((node_per_thread[par_thread] >= par) || (i == 0)){
            int cur_pos = sorted_index[i];
            int par_pos = sorted_index[par];
            for (int k = channel_idx * vertex_count; k < channel_size * vertex_count;
                       k += channel_step * vertex_count){
                float edge_weight = weight[i];
                out_data[cur_pos + k] = in_data[i + k] * (1 - edge_weight * edge_weight) +
                                        out_data[par_pos + k] * edge_weight;
                __threadfence_block();
            }
            node_per_thread[thread_idx] = i;
            i += thread_count;
        }
        __syncthreads();
    }
}

__global__ void leaf_root_aggr_kernel(
        float * in_data,
        float * out_data,
        float * weight,
        int * sorted_index,
        int * sorted_child_index,
        int batch_size,
        int channel_size,
        int vertex_count,
        int max_adj_per_node){

    const int thread_idx    = threadIdx.x;
    const int batch_idx     = blockIdx.x;
    const int channel_idx   = blockIdx.y;
    const int thread_count  = blockDim.x;
    const int channel_step  = gridDim.y;

    if (in_data != NULL){
        in_data    += batch_idx * vertex_count * channel_size;
    }
    out_data             += batch_idx * vertex_count * channel_size;
    weight               += batch_idx * vertex_count;
    sorted_index         += batch_idx * vertex_count;
    sorted_child_index   += batch_idx * vertex_count * max_adj_per_node;

    __shared__ int node_per_thread[CUDA_NUM_THREADS];
    node_per_thread[thread_idx] = vertex_count;
    __syncthreads();

    int i = vertex_count - thread_idx - 1;
    while (i >= 0){
        int child_len = 0;
        bool valid = true;
        for (int j = 0; j < max_adj_per_node; j++){
            int child        = sorted_child_index[i * max_adj_per_node + j];
            int child_thread = (vertex_count - child - 1) % thread_count;

            if (child <= 0) break;
            if (node_per_thread[child_thread] > child){
                valid = false;
                break;
            }
            child_len++;
        }
        if (valid){
            int cur_pos = sorted_index[i];
            for (int k = channel_idx * vertex_count; k < channel_size * vertex_count;
                    k += channel_step * vertex_count){
                float aggr_sum;
                if (in_data != NULL)
                    aggr_sum = in_data[cur_pos + k];
                else
                    aggr_sum = 1;
                for (int j = 0; j < child_len; j++){
                    int child = sorted_child_index[i * max_adj_per_node + j];
                    aggr_sum += out_data[child + k] * weight[child];
                }
                out_data[i + k] = aggr_sum;
            }
            node_per_thread[thread_idx] = i;
            i -= thread_count;
        }
        __syncthreads();
    }
}

__global__ void root_leaf_grad_kernel(
        float * in_data,
        float * in_grad,
        float * out_data,
        float * out_grad,
        float * weight,
        float * grad,
        int * sorted_index,
        int * sorted_parent_index,
        int batch_size,
        int data_channel_size,
        int grad_channel_size,
        int vertex_count){

    const int thread_idx    = threadIdx.x;
    const int batch_idx     = blockIdx.x;
    const int channel_idx   = blockIdx.y;
    const int thread_count  = blockDim.x;
    const int channel_step  = gridDim.y;
    const int channel_size  = data_channel_size > grad_channel_size ? data_channel_size : grad_channel_size;

    in_data             += batch_idx * vertex_count * data_channel_size;
    in_grad             += batch_idx * vertex_count * grad_channel_size;
    out_data            += batch_idx * vertex_count * data_channel_size;
    out_grad            += batch_idx * vertex_count * grad_channel_size;
    weight              += batch_idx * vertex_count;
    grad                += batch_idx * vertex_count * channel_size;
    sorted_index        += batch_idx * vertex_count;
    sorted_parent_index += batch_idx * vertex_count;

    __shared__ int node_per_thread[CUDA_NUM_THREADS];
    node_per_thread[thread_idx] = -1;

    int i = thread_idx;
    while (i < vertex_count){
        int cur         = i;
        int par         = sorted_parent_index[i];
        int par_pos     = sorted_index[par];
        int par_thread  = par % thread_count;
        if ((cur == 0) || (node_per_thread[par_thread] >= par)){
            for (int k = channel_idx; k < channel_size; k += channel_step){
                float edge_weight   = weight[i];
                int data_offset     = (k % data_channel_size) * vertex_count;
                int grad_offset     = (k % grad_channel_size) * vertex_count;
                int out_offset      = k * vertex_count;

                if (cur > 0){
                    float left  = in_grad[cur + grad_offset] * (out_data[par_pos + data_offset] - edge_weight * in_data[cur + data_offset]);
                    float right = in_data[cur + data_offset] * (out_grad[par + grad_offset] - edge_weight * in_grad[cur + grad_offset]);

                    grad[cur + out_offset]      = left + right;
                    out_grad[cur + grad_offset] = in_grad[cur + grad_offset] * (1 - edge_weight * edge_weight) +
                                                  out_grad[par + grad_offset] * edge_weight;
                    __threadfence_block();
                }
                else
                    grad[cur + out_offset] = 0;
            }
            node_per_thread[thread_idx] = i;
            i += thread_count;
        }
        __syncthreads();
    }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor>
tree_filter_refine_forward(
        const at::Tensor & feature_in_tensor,
        const at::Tensor & edge_weight_tensor,
        const at::Tensor & self_weight_tensor,
        const at::Tensor & sorted_index_tensor,
        const at::Tensor & sorted_parent_tensor,
        const at::Tensor & sorted_child_tensor
    ){

    const int batch_size        = feature_in_tensor.size(0);
    const int channel_size      = feature_in_tensor.size(1);
    const int vertex_size       = feature_in_tensor.size(2);
    const int max_adj_per_node  = sorted_child_tensor.size(2);

    auto options                  = feature_in_tensor.options();
    auto feature_aggr_tensor      = at::zeros_like(feature_in_tensor, options);
    auto feature_aggr_up_tensor   = at::zeros_like(feature_in_tensor, options);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    float * feature_in          = feature_in_tensor.contiguous().data_ptr<float>();
    float * edge_weight         = edge_weight_tensor.contiguous().data_ptr<float>();
    int * sorted_index          = sorted_index_tensor.contiguous().data_ptr<int>();
    int * sorted_parent_index   = sorted_parent_tensor.contiguous().data_ptr<int>();
    int * sorted_child_index    = sorted_child_tensor.contiguous().data_ptr<int>();
    float * feature_aggr        = feature_aggr_tensor.contiguous().data_ptr<float>();
    float * feature_aggr_sum    = feature_aggr_up_tensor.contiguous().data_ptr<float>();

    dim3 feature_block_dims(CUDA_NUM_THREADS, 1, 1), feature_grid_dims(batch_size, channel_size, 1);
    leaf_root_aggr_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
            feature_in, feature_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, channel_size, vertex_size, max_adj_per_node);
    root_leaf_prop_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
            feature_aggr_sum, feature_aggr, edge_weight, sorted_index, sorted_parent_index, batch_size, channel_size, vertex_size);

    auto feature_out_tensor = feature_aggr_tensor + (self_weight_tensor - 1).unsqueeze(1) * feature_in_tensor;

    auto result = std::make_tuple(feature_out_tensor, feature_aggr_tensor, feature_aggr_up_tensor);

    return result;
}

at::Tensor tree_filter_refine_backward_feature(
        const at::Tensor & feature_in_tensor,
        const at::Tensor & edge_weight_tensor,
        const at::Tensor & self_weight_tensor,
        const at::Tensor & sorted_index_tensor,
        const at::Tensor & sorted_parent_tensor,
        const at::Tensor & sorted_child_tensor,
        const at::Tensor & feature_aggr_tensor,
        const at::Tensor & feature_aggr_up_tensor,
        const at::Tensor & grad_out_tensor
    ){

    auto options                        = feature_in_tensor.options();
    auto grad_feature_tensor            = at::zeros_like(feature_in_tensor, options);
    auto grad_feature_aggr_sum_tensor   = at::zeros_like(feature_in_tensor, options);

    const int batch_size        = feature_in_tensor.size(0);
    const int channel_size      = feature_in_tensor.size(1);
    const int vertex_size       = feature_in_tensor.size(2);
    const int max_adj_per_node  = sorted_child_tensor.size(2);

    float * feature_in          = feature_in_tensor.contiguous().data_ptr<float>();
    float * edge_weight         = edge_weight_tensor.contiguous().data_ptr<float>();
    int * sorted_index          = sorted_index_tensor.contiguous().data_ptr<int>();
    int * sorted_parent_index   = sorted_parent_tensor.contiguous().data_ptr<int>();
    int * sorted_child_index    = sorted_child_tensor.contiguous().data_ptr<int>();
    float * feature_aggr        = feature_aggr_tensor.contiguous().data_ptr<float>();
    float * feature_aggr_sum    = feature_aggr_up_tensor.contiguous().data_ptr<float>();
    float * grad_out            = grad_out_tensor.contiguous().data_ptr<float>();
    float * grad_feature        = grad_feature_tensor.contiguous().data_ptr<float>();

    float * grad_feature_aggr_sum   = grad_feature_aggr_sum_tensor.contiguous().data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    dim3 feature_block_dims(CUDA_NUM_THREADS, 1, 1), feature_grid_dims(batch_size, channel_size, 1);
    leaf_root_aggr_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
            grad_out, grad_feature_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, channel_size, vertex_size, max_adj_per_node);
    root_leaf_prop_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
            grad_feature_aggr_sum, grad_feature, edge_weight, sorted_index, sorted_parent_index, batch_size, channel_size, vertex_size);

    grad_feature_tensor += (self_weight_tensor - 1).unsqueeze(1) * grad_out_tensor;

    return grad_feature_tensor;
}

at::Tensor tree_filter_refine_backward_edge_weight(
        const at::Tensor & feature_in_tensor,
        const at::Tensor & edge_weight_tensor,
        const at::Tensor & self_weight_tensor,
        const at::Tensor & sorted_index_tensor,
        const at::Tensor & sorted_parent_tensor,
        const at::Tensor & sorted_child_tensor,
        const at::Tensor & feature_aggr_tensor,
        const at::Tensor & feature_aggr_up_tensor,
        const at::Tensor & grad_out_tensor
    ){

    auto options            = feature_in_tensor.options();
    auto grad_weight_tensor = at::zeros_like(edge_weight_tensor, options);

    const int batch_size        = feature_in_tensor.size(0);
    const int channel_size      = feature_in_tensor.size(1);
    const int vertex_size       = feature_in_tensor.size(2);
    const int max_adj_per_node  = sorted_child_tensor.size(2);

    float * feature_in          = feature_in_tensor.contiguous().data_ptr<float>();
    float * edge_weight         = edge_weight_tensor.contiguous().data_ptr<float>();
    int * sorted_index          = sorted_index_tensor.contiguous().data_ptr<int>();
    int * sorted_parent_index   = sorted_parent_tensor.contiguous().data_ptr<int>();
    int * sorted_child_index    = sorted_child_tensor.contiguous().data_ptr<int>();
    float * feature_aggr        = feature_aggr_tensor.contiguous().data_ptr<float>();
    float * feature_aggr_sum    = feature_aggr_up_tensor.contiguous().data_ptr<float>();
    float * grad_out            = grad_out_tensor.contiguous().data_ptr<float>();
    float * grad_weight         = grad_weight_tensor.contiguous().data_ptr<float>();

    auto grad_all_channel_tensor        = at::zeros_like(feature_in_tensor, options);
    auto grad_norm_all_channel_tensor   = at::zeros_like(feature_in_tensor, options);
    auto grad_out_norm_aggr_sum_tensor  = at::zeros_like(feature_in_tensor, options);
    auto feature_grad_aggr_sum_tensor   = at::zeros_like(feature_in_tensor, options);

    float * grad_all_channel            = grad_all_channel_tensor.contiguous().data_ptr<float>();
    float * grad_norm_all_channel       = grad_norm_all_channel_tensor.contiguous().data_ptr<float>();
    float * grad_out_norm_aggr_sum      = grad_out_norm_aggr_sum_tensor.contiguous().data_ptr<float>();
    float * feature_grad_aggr_sum       = feature_grad_aggr_sum_tensor.contiguous().data_ptr<float>();

    auto grad_out_norm_tensor = grad_out_tensor;
    float * grad_out_norm     = grad_out_norm_tensor.contiguous().data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    dim3 feature_block_dims(CUDA_NUM_THREADS, 1, 1), feature_grid_dims(batch_size, channel_size, 1);
    leaf_root_aggr_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
            grad_out_norm, grad_out_norm_aggr_sum, edge_weight, sorted_index, sorted_child_index, batch_size, channel_size, vertex_size, max_adj_per_node);

    root_leaf_grad_kernel <<< feature_grid_dims, feature_block_dims, sizeof(int) * CUDA_NUM_THREADS, stream >>>(
            feature_aggr_sum, grad_out_norm_aggr_sum, feature_aggr, grad_out_norm_aggr_sum, edge_weight, grad_all_channel,
            sorted_index, sorted_parent_index, batch_size, channel_size, channel_size, vertex_size);

    grad_weight_tensor = grad_all_channel_tensor.sum(1);

    return grad_weight_tensor;
}

at::Tensor tree_filter_refine_backward_self_weight(
        const at::Tensor & feature_in_tensor,
        const at::Tensor & edge_weight_tensor,
        const at::Tensor & self_weight_tensor,
        const at::Tensor & sorted_index_tensor,
        const at::Tensor & sorted_parent_tensor,
        const at::Tensor & sorted_child_tensor,
        const at::Tensor & feature_aggr_tensor,
        const at::Tensor & feature_aggr_up_tensor,
        const at::Tensor & grad_out_tensor
    ){

    auto grad_self_weight_tensor = (grad_out_tensor * feature_in_tensor).sum(1);

    return grad_self_weight_tensor;
}
